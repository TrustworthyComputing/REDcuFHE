/**
 * Copyright (c) 2022 TrustworthyComputing - Charles Gouert
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/redcufhe_gpu.cuh>
#include <include/details/error_gpu.cuh>

using namespace redcufhe;

#include <omp.h>
#include <stdlib.h>
#include <time.h>
#include <utility>
#include <vector>
#include <math.h>
#include <iostream>
#include <ctime>
#include <ratio>
#include <chrono>
using namespace std;
using namespace std::chrono;

PriKey pri_key;
uint32_t kNumTests;
PubKey bk;

// shared vector used to issue/receive commands
vector<vector<pair<int, int>>> requests;


void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void setup(uint32_t kNumSMs, Ctxt** inputs, Ptxt** pt, Stream** st, int idx) {
  hipSetDevice(idx);

  // send bootstrapping key to GPU
  Initialize(bk); 

  // create CUDA streams for the GPU
  st[idx] = new Stream[kNumSMs]; 
  for (int i = 0; i < kNumSMs; i++) {
    st[idx][i].Create();
  }
  Synchronize();

  // Allocate memory for ciphertexts and encrypt
  (*inputs) = new Ctxt[2 * kNumTests];
  for (int i = 0; i < 2 * kNumTests; i++) {
    Encrypt((*inputs)[i], pt[idx][i], pri_key);
  }
  Synchronize();
  return;
}

// Runs on a worker CPU thread controlling a GPU
void server(int shares, uint32_t kNumSMs, int idx, Ctxt** answers, Stream** st) {
  while(1) {
    for (int i = 0; i < shares; i++) {
      // check for assignment
      if (requests[idx][i].first != -1) { 
        // terminate upon kill signal (-2)
        if (requests[idx][i].first == -2) { 
          Synchronize(); 
          return; 
        }
        // Perform gate computations
        Nand((*answers)[requests[idx][i].second], (*answers)[requests[idx][i].second], (*answers)[requests[idx][i].first], st[idx][i % kNumSMs]);
        // clear assignment
        requests[idx][i].first = -1; 
        requests[idx][i].second = -1;
      }
    }
  }
}

int main() {
  srand(time(NULL));

  // get GPU stats (WARNING: assumes all GPUs have the same number of SMs)
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  kNumTests = kNumSMs*kNumSMs*8; 

  // get number of available GPUs
  int numGPUs = 0;
  hipGetDeviceCount(&numGPUs);

  // create 2D array of plaintext and streams
  Ptxt* pt[numGPUs];
  Stream* st[numGPUs];

  // generate keyset
  SetSeed();
  PriKeyGen(pri_key);
  PubKeyGen(bk, pri_key);

  for (int i = 0; i < numGPUs; i++) {
  // generate random ptxts (bits) for each GPU
    pt[i] = new Ptxt[2 * kNumTests];
    for (int j = 0; j < 2 * kNumTests; j++) {
      pt[i][j] = rand() % Ptxt::kPtxtSpace;
    }
  }

  // Initialize shared vector for thread communication
  int num_threads = numGPUs;
  requests.resize(num_threads);
  for (int i = 0; i < num_threads; i++) {
    requests[i].resize(kNumTests);
    for (int j = 0; j < kNumTests; j++) {
      // each element holds indices of data array
      requests[i][j] = make_pair(-1,-1);
    }
  }

  Ctxt* answers[numGPUs];
  omp_set_num_threads(numGPUs);

  // Initialize data on each available GPU
  #pragma omp parallel for shared(st, answers)
  for (int i = 0; i < numGPUs; i++) {
    setup(kNumSMs, &answers[i], pt, st, i);
  }

  // one worker thread for each GPU and a scheduler thread
  omp_set_num_threads(numGPUs+1);


  high_resolution_clock::time_point t1 = high_resolution_clock::now();

  /////////////////////////////////////////
  //
  // (RED)cuFHE Dynamic Scheduler
  // Enables automatic allocation of FHE 
  // workloads to multiple GPUs
  //
  //////////////////////////////////////////   
  #pragma omp parallel for shared(answers, st, requests)
  for (int i = 0; i < (num_threads+1); i++) {
    if (i != 0) { // workers
      int thread_id = omp_get_thread_num() - 1;
      hipSetDevice(thread_id);
      server(kNumTests, kNumSMs, thread_id, &answers[i-1], st);
      Synchronize();
    }
    else { // master thread
      int turn = 1; // indicates target worker 
      for (int j = 0; j < (kNumTests*numGPUs); j++) {
        if ((j % kNumTests == 0) && (j > 0)) {
          turn++; // assign to next worker
          if (turn > num_threads) { // excludes scheduler
            turn = 1;
          }
        }
        // assign input 1 as index j of GPU array
        requests[turn-1][j % kNumTests].second = j % (kNumTests);
        // assign input 2 as index j+kNumTests
        requests[turn-1][j % kNumTests].first = ((j%kNumTests)+kNumTests) % (2*kNumTests);
      }
      // check to see if all threads are done
      bool end = false;
      while (end == false) {
        end = true;
        for (int j = 0; j < num_threads; j++) {
          for (int k = 0; k < kNumTests; k++) {
            if (requests[j][k].first != -1) {
              end = false;
              break;
            }
          }
        }
      }
      // terminate workers
      for (int j = 0; j < num_threads; j++) {
        for (int k = 0; k < kNumTests; k++) {
          requests[j][k].first = -2;
        }
      }
    }
  }

  cout << "Gate evals: " << kNumTests*numGPUs << endl;
  
  // Confirm results and check for errors
  int wrong_counter[numGPUs];
  omp_set_num_threads(numGPUs);
  #pragma omp parallel shared(wrong_counter)
  {
    Ptxt* recovered_pt = new Ptxt[kNumTests];
    int thread_num = omp_get_thread_num();
    hipSetDevice(thread_num);
    for (int i = 0; i < kNumTests; i++) {
      NandCheck(pt[thread_num][i], pt[thread_num][i+kNumTests], pt[thread_num][i]);
      Decrypt(recovered_pt[i], answers[thread_num][i+kNumTests], pri_key);
    }
    wrong_counter[thread_num] = 0;
    for (int i = 0; i < kNumTests; i++) {
      if (pt[thread_num][i+kNumTests].message_ != recovered_pt[i].message_) {
        wrong_counter[thread_num]++;
      }
    }
    delete [] recovered_pt;
  }

  for (int i = 0; i < numGPUs; i++) {
    cout << "GPU #" << i << " errors: " << wrong_counter[i] << endl;
  }

  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
  std::cout << "Time: " << time_span.count() << " seconds" << endl;

  for (int i = 0; i < numGPUs; i++) {
    delete [] pt[i];
  }
  // free GPU memory
  CleanUp();

  return 0;
}


