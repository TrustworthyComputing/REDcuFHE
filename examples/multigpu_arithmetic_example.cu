/**
 * Copyright (c) 2022 TrustworthyComputing - Charles Gouert
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

//
// Multi-GPU Leveled Arithmetic Operations example
//
// This example highlights the new leveled
// features in the integer domain. Instead
// of operating over encrypted bits, which
// is required in cuFHE, (RED)cuFHE allows
// for operations over encrypted modular integers.
//

#include <include/redcufhe_gpu.cuh>
#include <include/details/error_gpu.cuh>

using namespace redcufhe;

#include <omp.h>
#include <stdlib.h>
#include <time.h>
#include <utility>
#include <vector>
#include <math.h>
#include <iostream>
#include <ctime>
#include <ratio>
#include <chrono>
using namespace std;
using namespace std::chrono;

PriKey pri_key;
uint32_t kNumTests;
PubKey bk;

// shared vector used to issue/receive commands
vector<vector<pair<int, int>>> requests;

// plaintext modulus
uint32_t message_space = 256;

void AddCheck(int32_t& out, const int32_t& in0, const int32_t& in1) {
  out = in0 + in1;
}

void setup(uint32_t kNumSMs, Ctxt** inputs, int32_t** pt, Stream** st, int idx) {
  hipSetDevice(idx);

  // send bootstrapping key to GPU
  Initialize(bk);

  // create CUDA streams for the GPU
  st[idx] = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i++) {
    st[idx][i].Create();
  }
  Synchronize();

  // Allocate memory for ciphertexts and encrypt
  (*inputs) = new Ctxt[2 * kNumTests];
  for (int i = 0; i < 2 * kNumTests; i++) {
    EncryptIntRed((*inputs)[i], pt[idx][i], message_space, pri_key);
  }
  Synchronize();
  return;
}

// Runs on a worker CPU thread controlling a GPU
void server(int shares, uint32_t kNumSMs, int idx, Ctxt** answers, Stream** st) {
  while(1) {
    for (int i = 0; i < shares; i++) {
      // check for assignment
      if (requests[idx][i].first != -1) {
        // terminate upon kill signal (-2)
        if (requests[idx][i].first == -2) {
          Synchronize();
          return;
        }
        // Perform leveled addition
        AddRed((*answers)[requests[idx][i].second], (*answers)[requests[idx][i].second], (*answers)[requests[idx][i].first], st[idx][i % kNumSMs]);
        // clear assignment
        requests[idx][i].first = -1;
        requests[idx][i].second = -1;
      }
    }
  }
}

int main() {
  srand(time(NULL));

  // get GPU stats (WARNING: assumes all GPUs have the same number of SMs)
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  kNumTests = kNumSMs*kNumSMs*8;// * 8;

  // get number of available GPUs
  int numGPUs = 0;
  hipGetDeviceCount(&numGPUs);

  // create 2D array of plaintext and streams
  int32_t* pt[numGPUs]; // int is used instead of binary Ptxt
  Stream* st[numGPUs];

  // generate keyset
  SetSeed();
  PriKeyGen(pri_key);
  PubKeyGen(bk, pri_key);

  for (int i = 0; i < numGPUs; i++) {
  // generate random ptxts (integers) for each GPU
    pt[i] = new int32_t[2 * kNumTests];
    for (int j = 0; j < 2 * kNumTests; j++) {
      pt[i][j] = rand() % (message_space / 2);
    }
  }

  // Initialize shared vector for thread communication
  int num_threads = numGPUs;
  requests.resize(num_threads);
  for (int i = 0; i < num_threads; i++) {
    requests[i].resize(kNumTests);
    for (int j = 0; j < kNumTests; j++) {
      // each element holds indices of data array
      requests[i][j] = make_pair(-1,-1);
    }
  }

  Ctxt* answers[numGPUs];
  omp_set_num_threads(numGPUs);

  // Initialize data on each available GPU
  #pragma omp parallel for shared(st, answers)
  for (int i = 0; i < numGPUs; i++) {
    setup(kNumSMs, &answers[i], pt, st, i);
  }

  // one worker thread for each GPU and a scheduler thread
  omp_set_num_threads(numGPUs+1);

  high_resolution_clock::time_point t1 = high_resolution_clock::now();

  /////////////////////////////////////////
  //
  // (RED)cuFHE Dynamic Scheduler
  // Enables automatic allocation of FHE
  // workloads to multiple GPUs
  //
  /////////////////////////////////////////
  #pragma omp parallel for shared(answers, st, requests)
  for (int i = 0; i < (num_threads+1); i++) {
    if (i != 0) { // workers
      int thread_id = omp_get_thread_num() - 1;
      hipSetDevice(thread_id);
      server(kNumTests, kNumSMs, thread_id, &answers[i-1], st);
      Synchronize();
    }
    else { // master thread
      int turn = 1; // indicates target worker
      for (int j = 0; j < (kNumTests*numGPUs); j++) {
        if ((j % kNumTests == 0) && (j > 0)) {
          turn++; // assign to next worker
          if (turn > num_threads) { // excludes scheduler
            turn = 1;
          }
        }
        // assign input 1 as index j of GPU array
        requests[turn-1][j % kNumTests].second = j % (kNumTests);
        // assign input 2 as index j+kNumTests
        requests[turn-1][j % kNumTests].first = ((j%kNumTests)+kNumTests) % (2*kNumTests);
      }
      // check to see if all threads are done
      bool end = false;
      while (end == false) {
        end = true;
        for (int j = 0; j < num_threads; j++) {
          for (int k = 0; k < kNumTests; k++) {
            if (requests[j][k].first != -1) {
              end = false;
              break;
            }
          }
        }
      }
      // terminate workers
      for (int j = 0; j < num_threads; j++) {
        for (int k = 0; k < kNumTests; k++) {
          requests[j][k].first = -2;
        }
      }
    }
  }

  cout << "Arithmetic evals: " << kNumTests*numGPUs << endl;

  // Confirm results and check for errors
  int wrong_counter[numGPUs];
  omp_set_num_threads(numGPUs);
  #pragma omp parallel shared(wrong_counter)
  {
    int32_t* recovered_pt = new int32_t[kNumTests];
    int thread_num = omp_get_thread_num();
    hipSetDevice(thread_num);
    for (int i = 0; i < kNumTests; i++) {
      AddCheck(pt[thread_num][i], pt[thread_num][i+kNumTests], pt[thread_num][i]);
      DecryptIntRed(recovered_pt[i], answers[thread_num][i+kNumTests], message_space, pri_key);
    }
    wrong_counter[thread_num] = 0;
    for (int i = 0; i < kNumTests; i++) {
      if (pt[thread_num][i+kNumTests] != recovered_pt[i]) {
        cout << "Expected: " << pt[thread_num][i+kNumTests] << "  Actual: " << recovered_pt[i] << endl;
        wrong_counter[thread_num]++;
      }
    }
    delete [] recovered_pt;
  }

  for (int i = 0; i < numGPUs; i++) {
    cout << "GPU #" << i << " errors: " << wrong_counter[i] << endl;
  }

  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
  std::cout << "Time: " << time_span.count() << " seconds" << endl;

  for (int i = 0; i < numGPUs; i++) {
    delete [] pt[i];
  }
  // free GPU memory
  CleanUp();

  return 0;
}
