#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2022 TrustworthyComputing - Charles Gouert
 * 
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */


#include <include/redcufhe_bootstrap_gpu.cuh>
#include <include/ntt_gpu/ntt.cuh>
#include <include/details/error_gpu.cuh>

#include <iostream>
using namespace std;

#define NUM_GPUS 8 //FIXME: Adjust as needed

namespace redcufhe {

using BootstrappingKeyNTT = TGSWSampleArray_T<FFP>;
BootstrappingKeyNTT* bk_ntt[NUM_GPUS] = {nullptr};
MemoryDeleter bk_ntt_deleter[NUM_GPUS] = {nullptr};
KeySwitchingKey* ksk_dev[NUM_GPUS] = {nullptr};
MemoryDeleter ksk_dev_deleter[NUM_GPUS] = {nullptr};
CuNTTHandler<>* ntt_handler[NUM_GPUS] = {nullptr};

__global__ void __BootstrappingKeyToNTT__(BootstrappingKeyNTT bk_ntt,
                                          BootstrappingKey bk,
                                          CuNTTHandler<> ntt) {
  __shared__ FFP sh_temp[1024];

  TGSWSample tgsw;
  bk.ExtractTGSWSample(&tgsw, blockIdx.z);
  TLWESample tlwe;
  tgsw.ExtractTLWESample(&tlwe, blockIdx.y);
  Torus* poly_in = tlwe.ExtractPoly(blockIdx.x);

  TGSWSample_T<FFP> tgsw_ntt;
  bk_ntt.ExtractTGSWSample(&tgsw_ntt, blockIdx.z);
  TLWESample_T<FFP> tlwe_ntt;
  tgsw_ntt.ExtractTLWESample(&tlwe_ntt, blockIdx.y);
  FFP* poly_out = tlwe_ntt.ExtractPoly(blockIdx.x);
  ntt.NTT<Torus>(poly_out, poly_in, sh_temp, 0);
}

void BootstrappingKeyToNTT(const BootstrappingKey* bk) {
  int idx;
  hipGetDevice(&idx);
  BootstrappingKey* d_bk;
  d_bk = new BootstrappingKey(bk->n(), bk->k(), bk->l(), bk->w(), bk->t());
  std::pair<void*, MemoryDeleter> pair;
  pair = AllocatorGPU::New(d_bk->SizeMalloc());
  d_bk->set_data((BootstrappingKey::PointerType)pair.first);
  MemoryDeleter d_bk_deleter = pair.second;
  CuSafeCall(hipMemcpy(d_bk->data(), bk->data(), d_bk->SizeMalloc(),
                        hipMemcpyHostToDevice));

  if (bk_ntt[idx] == nullptr) {
     bk_ntt[idx] = new BootstrappingKeyNTT(bk->n(), bk->k(), bk->l(), bk->w(), bk->t());
  }
  pair = AllocatorGPU::New(bk_ntt[idx]->SizeMalloc());
  bk_ntt[idx]->set_data((BootstrappingKeyNTT::PointerType)pair.first);
  bk_ntt_deleter[idx] = pair.second;
  //Assert(ntt_handler == nullptr);
  ntt_handler[idx] = new CuNTTHandler<>();
  ntt_handler[idx]->Create();
  ntt_handler[idx]->CreateConstant();
  hipDeviceSynchronize();
  CuCheckError();
  dim3 grid(bk->k() + 1, (bk->k() + 1) * bk->l(), bk->t());
  dim3 block(128);
  __BootstrappingKeyToNTT__<<<grid, block>>>(*(bk_ntt[idx]), *d_bk, *ntt_handler[idx]);
  hipDeviceSynchronize();
  CuCheckError();

  d_bk_deleter(d_bk->data());
  delete d_bk;
}

void DeleteBootstrappingKeyNTT() {
  int idx;
  hipGetDevice(&idx);
  bk_ntt_deleter[idx](bk_ntt[idx]->data());
  delete  bk_ntt[idx];
  bk_ntt[idx] = nullptr;

  ntt_handler[idx]->Destroy();
  delete  ntt_handler[idx];
}

void KeySwitchingKeyToDevice(const KeySwitchingKey* ksk) {
  //Assert(ksk_dev == nullptr);
  int idx;
  hipGetDevice(&idx);
  ksk_dev[idx] = new KeySwitchingKey(ksk->n(), ksk->l(), ksk->w(), ksk->m());
  std::pair<void*, MemoryDeleter> pair;
  pair = AllocatorGPU::New(ksk_dev[idx]->SizeMalloc());
  ksk_dev[idx]->set_data((KeySwitchingKey::PointerType)pair.first);
  ksk_dev_deleter[idx] = pair.second;
  CuSafeCall(hipMemcpy(ksk_dev[idx]->data(), ksk->data(), ksk->SizeMalloc(),
                        hipMemcpyHostToDevice));
}

void DeleteKeySwitchingKey() {
  int idx;
  hipGetDevice(&idx);
  ksk_dev_deleter[idx](ksk_dev[idx]->data());
  delete  ksk_dev[idx];
  ksk_dev[idx] = nullptr;
}

__device__ inline
uint32_t ModSwitch2048(uint32_t a) {
  return (((uint64_t)a << 32) + (0x1UL << 52)) >> 53;
}

template <uint32_t lwe_n = 500, uint32_t tlwe_n = 1024,
          uint32_t decomp_bits = 2, uint32_t decomp_size = 8>
__device__ inline
void KeySwitch(Torus* lwe, Torus* tlwe, Torus* ksk) {
  static const Torus decomp_mask = (1u << decomp_bits) - 1;
  static const Torus decomp_offset = 1u << (31 - decomp_size * decomp_bits);
  uint32_t tid = ThisThreadRankInBlock();
  uint32_t bdim = ThisBlockSize();
  Torus tmp;
  Torus res = 0;
  Torus val = 0;
  #pragma unroll 0
  for (int i = tid; i <= lwe_n; i += bdim) {
    if (i == lwe_n)
      res = tlwe[tlwe_n];
    #pragma unroll 0
    for (int j = 0; j < tlwe_n; j ++) {
      if (j == 0)
        tmp = tlwe[0];
      else
        tmp = -tlwe[1024 - j];
      tmp += decomp_offset;
      for (int k = 0; k < decomp_size; k ++) {
        val = (tmp >> (32 - (k + 1) * decomp_bits)) & decomp_mask;
        if (val != 0)
          res -= ksk[(j << 14) | (k << 11) | (val << 9) | i];
      }
    }
    lwe[i] = res;
  }
}

__device__
void Accumulate(Torus* tlwe,
                FFP* sh_acc_ntt,
                FFP* sh_res_ntt,
                uint32_t a_bar,
                FFP* tgsw_ntt,
                CuNTTHandler<> ntt) {
  static const uint32_t decomp_bits = 10;
  static const uint32_t decomp_mask = (1 << decomp_bits) - 1;
  static const int32_t decomp_half = 1 << (decomp_bits - 1);
  static const uint32_t decomp_offset = (0x1u << 31) +
                                        (0x1u << (31 - decomp_bits));
  uint32_t tid = ThisThreadRankInBlock();
  uint32_t bdim = ThisBlockSize();

  // temp[2] = sh_acc[2] * (x^exp - 1)
  // sh_acc_ntt[0, 1] = Decomp(temp[0])
  // sh_acc_ntt[2, 3] = Decomp(temp[1])
  // This algorithm is tested in cpp.
  Torus temp;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    uint32_t cmp = (uint32_t)(i < (a_bar & 1023));
    uint32_t neg = -(cmp ^ (a_bar >> 10));
    uint32_t pos = -((1 - cmp) ^ (a_bar >> 10));
    #pragma unroll
    for (int j = 0; j < 2; j ++) {
      temp = tlwe[(j << 10) | ((i - a_bar) & 1023)];
      temp = (temp & pos) + ((-temp) & neg);
      temp -= tlwe[(j << 10) | i];
      // decomp temp
      temp += decomp_offset;
      sh_acc_ntt[(2*j)*1024+i] = FFP(Torus( ((temp >> (32 - decomp_bits))
                             & decomp_mask) - decomp_half ));
      sh_acc_ntt[(2*j+1)*1024+i] = FFP(Torus( ((temp >> (32 - 2 * decomp_bits))
                                 & decomp_mask) - decomp_half ));
    }
  }
  __syncthreads(); // must

  // 4 NTTs with 512 threads.
  // Input/output/buffer use the same shared memory location.
  if (tid < 512) {
    FFP* tar = &sh_acc_ntt[tid >> 7 << 10];
    ntt.NTT<FFP>(tar, tar, tar, tid >> 7 << 7);
  }
  else { // must meet 4 sync made by NTTInv
    __syncthreads();
    __syncthreads();
    __syncthreads();
    __syncthreads();
  }
  __syncthreads();

  // Multiply with bootstrapping key in global memory.
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    sh_res_ntt[4096+i] = 0;
    #pragma unroll
    for (int j = 0; j < 4; j ++)
      sh_res_ntt[4096+i] += sh_acc_ntt[j*1024+i] * tgsw_ntt[((2 * j + 1) << 10) + i];
  }
  __syncthreads(); // new
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    FFP temp = 0;
    #pragma unroll
    for (int j = 0; j < 4; j ++)
      temp += sh_acc_ntt[j*1024+i] * tgsw_ntt[((2 * j) << 10) + i];
    sh_res_ntt[i] = temp;
  }
  __syncthreads(); // must

  // 2 NTTInvs and add acc with 256 threads.
  if (tid < 256) {
    FFP* src = &sh_res_ntt[tid >> 7 << 12];
    ntt.NTTInvAdd<Torus>(&tlwe[tid >> 7 << 10], src, src, tid >> 7 << 7);
  }
  else { // must meet 4 sync made by NTTInv
    __syncthreads();
    __syncthreads();
    __syncthreads();
    __syncthreads();
  }
  __syncthreads(); // must
}

__global__
void __Bootstrap__(Torus* out, Torus* in, Torus mu,
                   FFP* bk,
                   Torus* ksk,
                   CuNTTHandler<> ntt) {
//  Assert(bk.k() == 1);
//  Assert(bk.l() == 2);
//  Assert(bk.n() == 1024);
  __shared__ FFP sh[6 * 1024];
//  FFP* sh_acc_ntt[4] = { sh, sh + 1024, sh + 2048, sh + 3072 };
//  FFP* sh_res_ntt[2] = { sh, sh + 4096 };
  Torus* tlwe = (Torus*)&sh[5120];

  // test vector
  // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(in[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(in[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }

  static const uint32_t lwe_n = 500;
  static const uint32_t tlwe_n = 1024;
  static const uint32_t ks_bits = 2;
  static const uint32_t ks_size = 8;
  KeySwitch<lwe_n, tlwe_n, ks_bits, ks_size>(out, tlwe, ksk);
}

__global__
void __NandBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu, Torus fix,
                       FFP* bk, Torus* ksk, CuNTTHandler<> ntt) {
  __shared__ FFP sh[6 * 1024];
  Torus* tlwe = (Torus*)&sh[5120];
  // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(fix - in0[500] - in1[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(0 - in0[i] - in1[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }
  KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__
void __OrBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu, Torus fix,
                       FFP* bk, Torus* ksk, CuNTTHandler<> ntt) {
  __shared__ FFP sh[6 * 1024];
  Torus* tlwe = (Torus*)&sh[5120];
  // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(fix + in0[500] + in1[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(0 + in0[i] + in1[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }
  KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__
void __AndBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu, Torus fix,
                       FFP* bk, Torus* ksk, CuNTTHandler<> ntt) {
  __shared__ FFP sh[6 * 1024];
  Torus* tlwe = (Torus*)&sh[5120];
  // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(fix + in0[500] + in1[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(0 + in0[i] + in1[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }
  KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__
void __NorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu, Torus fix,
                       FFP* bk, Torus* ksk, CuNTTHandler<> ntt) {
  __shared__ FFP sh[6 * 1024];
  Torus* tlwe = (Torus*)&sh[5120];
  // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(fix - in0[500] - in1[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(0 - in0[i] - in1[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }
  KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__
void __XorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu, Torus fix,
                       FFP* bk, Torus* ksk, CuNTTHandler<> ntt) {
  __shared__ FFP sh[6 * 1024];
  Torus* tlwe = (Torus*)&sh[5120];
  // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(fix + 2*in0[500] + 2*in1[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(0 + 2*in0[i] + 2*in1[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }
  KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

__global__
void __XnorBootstrap__(Torus* out, Torus* in0, Torus* in1, Torus mu, Torus fix,
                       FFP* bk, Torus* ksk, CuNTTHandler<> ntt) {
  __shared__ FFP sh[6 * 1024];
  Torus* tlwe = (Torus*)&sh[5120];
  // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
  register int32_t bar = 2048 - ModSwitch2048(fix - 2*in0[500] - 2*in1[500]);
  register uint32_t tid = ThisThreadRankInBlock();
  register uint32_t bdim = ThisBlockSize();
  register uint32_t cmp, neg, pos;
  #pragma unroll
  for (int i = tid; i < 1024; i += bdim) {
    tlwe[i] = 0; // part a
    if (bar == 2048)
      tlwe[i + 1024] = mu;
    else {
      cmp = (uint32_t)(i < (bar & 1023));
      neg = -(cmp ^ (bar >> 10));
      pos = -((1 - cmp) ^ (bar >> 10));
      tlwe[i + 1024] = (mu & pos) + ((-mu) & neg); // part b
    }
  }
  __syncthreads();
  // accumulate
  #pragma unroll
  for (int i = 0; i < 500; i ++) { // 500 iterations
    bar = ModSwitch2048(0 - 2*in0[i] - 2*in1[i]);
    Accumulate(tlwe, sh, sh, bar, bk + (i << 13), ntt);
  }
  KeySwitch<500, 1024, 2, 8>(out, tlwe, ksk);
}

void Bootstrap(LWESample* out,
               LWESample* in,
               Torus mu,
               hipStream_t st) {
  dim3 grid(1);
  dim3 block(512);
  int idx;
  hipGetDevice(&idx);
  __Bootstrap__<<<grid, block, 0, st>>>(out->data(), in->data(), mu,
      bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

void NandBootstrap(LWESample* out, LWESample* in0, LWESample* in1,
    Torus mu, Torus fix, hipStream_t st) {
  int idx;
  hipGetDevice(&idx);
  __NandBootstrap__<<<1, 512, 0, st>>>(out->data(), in0->data(),
      in1->data(), mu, fix, bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

void OrBootstrap(LWESample* out, LWESample* in0, LWESample* in1,
    Torus mu, Torus fix, hipStream_t st) {
  int idx;
  hipGetDevice(&idx);
  __OrBootstrap__<<<1, 512, 0, st>>>(out->data(), in0->data(),
      in1->data(), mu, fix, bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

void AndBootstrap(LWESample* out, LWESample* in0, LWESample* in1,
    Torus mu, Torus fix, hipStream_t st) {
  int idx;
  hipGetDevice(&idx);
  __AndBootstrap__<<<1, 512, 0, st>>>(out->data(), in0->data(),
      in1->data(), mu, fix, bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

void NorBootstrap(LWESample* out, LWESample* in0, LWESample* in1,
    Torus mu, Torus fix, hipStream_t st) {
  int idx;
  hipGetDevice(&idx);
  __NorBootstrap__<<<1, 512, 0, st>>>(out->data(), in0->data(),
      in1->data(), mu, fix, bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

void XorBootstrap(LWESample* out, LWESample* in0, LWESample* in1,
    Torus mu, Torus fix, hipStream_t st) {
  int idx;
  hipGetDevice(&idx);
  __XorBootstrap__<<<1, 512, 0, st>>>(out->data(), in0->data(),
      in1->data(), mu, fix, bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

void XnorBootstrap(LWESample* out, LWESample* in0, LWESample* in1,
    Torus mu, Torus fix, hipStream_t st) {
  int idx;
  hipGetDevice(&idx);
  __XnorBootstrap__<<<1, 512, 0, st>>>(out->data(), in0->data(),
      in1->data(), mu, fix, bk_ntt[idx]->data(), ksk_dev[idx]->data(), *ntt_handler[idx]);
  CuCheckError();
}

} // namespace redcufhe
